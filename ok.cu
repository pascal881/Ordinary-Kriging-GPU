#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>
#include <time.h>

#include "mylib.h"
#include "print.h"
#include "dist2.h"

#define MAXBUFF 4096

__global__ void computeMatrix(float* matrice,int m,point2d *kp)
{
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	int lda = m;
	
	
	if(i>=1 && i<m)
	{
	float w,s,d;
	
	int j = 1;
	
	for(j = 1; j < m ; j++)
	{
		int tempi,tempj,mm;
		tempi = i-1;
		tempj = j-1;

		mm = lda-1;

	

		d = distGPU(kp[i],kp[j]);
		s = 10;
		w = s * exp(-3*(d/10));
		
		
		//Computing the index for access to matrix (vector' format)
		matrice[(tempi*lda)+tempj] = w;
		
		matrice[(tempi*lda)+m-1] = 1;
		matrice[(mm*lda)+tempj] = 1;
		
	}
	
	}
}

__global__ void computeMatrixD(float *d_D,point2d* d_kp,int m)
{
	int i = threadIdx.x + (blockIdx.x * blockDim.x);

	if ( i >= 1 && i < m)
		{

		float d,w,s;
		s = 10;
		d = distGPU(d_kp[i],d_kp[0]);
		w = s * exp(-3 * (d/10));
	
		d_D[i-1] = w;

		__syncthreads();
		}

	// Only master thread
	if( i == 0 )
		{
			d_D[m-1] = 1; //Lagrange factor

		}
	if(i<m)

	__syncthreads();

}

__global__ void swapValue(point2d* d_kp,point2d* d_qp,int k)
{
	int i = threadIdx.x + (blockIdx.x * blockDim.x);


	//Master thread swap the #k point in queryPoint set to first location of knowPoints	
	if( i == 0 )
		{

			d_kp[i] = d_qp[k];

		}

		__syncthreads();		
}

int main(int argc,char* argv[])
{
	char path1[MAXBUFF],path2[MAXBUFF];
	
	int flag_save = 0; // Flag to choose if save the results

	//Data for time
	hipEvent_t start1,stop;

	point2d *kp,*qp;
	point2d *kpd,*qpd;

	float *z,*zfinal;
	float *zd,*zfinald;


	int i = 0;


	dim3 nb,nt;

	int n,m;

	float *matricegpu;
	//float *matricecpu;

	float *dd;

	/* CUSOLVER DATA */
	hipsolverHandle_t start;
	hipsolverStatus_t status;

	int *Lwork;
	float *Workspace;
	int *devIpiv; //Pivot LU
	int *devInfo;

	start = NULL;
	
	/* CUSOLVER DATA */
	int lda,ldb,nrhs;
	nrhs = 1;

	/* CUBLAS DATA */
	hipblasHandle_t handlet;
        hipblasStatus_t statusCub;

	if(argc>1)
		{
			flag_save = atoi(argv[1]);
			strcpy(path1,argv[2]);
			strcpy(path2,argv[3]);
			
			if(argc>4)
				nt.x = atoi(argv[4]);
		}
	else
		{
			printf("Error usage : ./exec <flagSave> <knownPointsfile> <queryPointsDataset>\n");
			exit(-1);
		}


	m = getLines(path1);
	n = getLines(path2);

//	m--;
	n--;		
	lda = m;
	ldb = m;


	if(argc<3)
	nt.x = m;

	kp = (point2d*)malloc(m*sizeof(point2d));
	qp = (point2d*)malloc(n*sizeof(point2d));
	z = (float*)malloc(m*sizeof(point2d));
	zfinal = (float*)malloc(n*sizeof(float));

	//Calcolo del numero di blocchi
	nb.x = 	m/nt.x + ( (m%nt.x) == 0 ? 0 : 1);

	printf("Flag save results : ");
	if(flag_save == 1 )
		printf("yes\n");
	else printf("no\n");

	printf("KnownPoints : %d\n",m);
	printf("QueryPoints : %d\n",n);
	printf("#Threads : %d\n",nt.x);
	printf("#Blocks : %d\n",nb.x);	
	
	/* APERTURA E LETTURA DA FILE */
	FILE *f1,*f2;
	f1 = fopen(path1,"r");
	
	for( i=0 ; i<m; i++)
		fscanf(f1,"%f;%f;%f\n",&kp[i+1].x,&kp[i+1].y,&z[i]);	
	fclose(f1);

	
	f2 = fopen(path2,"r");
	
	for(i=0;i<n;i++)
		fscanf(f2,"%f;%f;\n",&qp[i].x,&qp[i].y);
	
	fclose(f2);
/*
	printf("KnwonPoints :\nX\t\tY\t\tZ\n");
	for(i=0;i<m;i++)
		printf("%f\t%f\t%f\n",kp[i].x,kp[i].y,z[i]);
	
	printf("\nQueryPoints:\nX\t\tY\n");
	for(i=0;i<n;i++)
		printf("%f\t%f\n",qp[i].x,qp[i].y);
*/	
	hipEventCreate(&start1);
	hipEventCreate(&stop);

	//Allocazione dei vettori dei punti su GPU
	hipMalloc((void**)&kpd,m*sizeof(point2d));
	hipMalloc((void**)&qpd,n*sizeof(point2d));
	hipMalloc((void**)&zd,m*sizeof(float));
	hipMalloc((void**)&zfinald,n*sizeof(float));

	//Allocazione matrice 
	hipMalloc((void**)&matricegpu,m*m*sizeof(float));
	hipMalloc((void**)&dd,m*sizeof(float));

//	matricecpu = (float*)malloc(m*m*sizeof(float));
	//d = (float*)malloc(m*sizeof(float));

	//Copia dei vettori da CPU a GPU
	hipMemcpy(kpd,kp,m*sizeof(point2d),hipMemcpyHostToDevice);
	hipMemcpy(qpd,qp,n*sizeof(point2d),hipMemcpyHostToDevice);
	hipMemcpy(zd,z,m*sizeof(float),hipMemcpyHostToDevice);

	Lwork = (int*)malloc(sizeof(int));
	
	hipEventRecord(start1);

	computeMatrix<<<nb.x,nt.x>>>(matricegpu,m,kpd);

	//hipMemcpy(matricecpu,matricegpu,m*m*sizeof(float),hipMemcpyDeviceToHost);
	
	 

	//Creazione contensto CUSOLVER
	if( hipsolverDnCreate(&start) == HIPSOLVER_STATUS_SUCCESS)
		printf("CUSOLVER OK\n");	

	status = hipsolverDnSgetrf_bufferSize(start,m,m,matricegpu,lda,Lwork);
	
	//Calcolo buffersize
/*	if( status == HIPSOLVER_STATUS_SUCCESS)
		printf("BUFFERSIZE OK\n");	
	printf("Workspace : %d\n",(*Lwork));
*/
	//Allocazione su GPU del workspace
	hipMalloc((void**)&Workspace,sizeof(float)*(*Lwork));
	hipMalloc((void**)&devIpiv,sizeof(int)*m);
	hipMalloc((void**)&devInfo,sizeof(int));

	//Fattorizzazione
	status = hipsolverDnSgetrf(start,m,m,matricegpu,lda,Workspace,devIpiv,devInfo);
	if( status == HIPSOLVER_STATUS_SUCCESS)
		printf("LU Factorization ok\n");
	
	

	int k = 0;
	

	if(hipblasCreate(&handlet) == HIPBLAS_STATUS_SUCCESS)
		printf("CUBLAS OK\n");
	

	float o = 0;
	int incx = 1;
	int incy = 1;
	
	for(k=0;k<n;k++)
	{
	
	swapValue<<<1,1>>>(kpd,qpd,k);
	computeMatrixD<<<nb.x,nt.x>>>(dd,kpd,m);

		
	status = hipsolverDnSgetrs(start,HIPBLAS_OP_C,m,nrhs,matricegpu,lda,devIpiv,dd,ldb,devInfo);
	if(status != HIPSOLVER_STATUS_SUCCESS)
	{
		printf("Solver cusolver problem\n");
		exit(-1);
	}


	
//	float o;
//	int incx = 1;
//	int incy = 1;

	// solve the dot product between 2 vector and save the result in "o" variable
	statusCub = hipblasSdot(handlet,m-1,dd,incx,zd,incy,&o);
	if(statusCub != HIPBLAS_STATUS_SUCCESS)
		{
			printf("CUBLAS problem\n");
			exit(-1);
		}

	zfinal[k] = o;
	
	}

	hipEventRecord(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds,start1,stop);
	printf("GPU Time elapsed : %f s\n",milliseconds*0.001);

	//Copia della matrice creata , viene copiata dalla GPU alla CPU per debug
	//hipMemcpy(matricecpu,matricegpu,m*m*sizeof(float),hipMemcpyDeviceToHost);
	
/*	printf("Valori finali: \nX\t\tY\t\tZ\n");
	for(i=0;i<n;i++)
		printf("%f\t%f\t%f\n",qp[i].x,qp[i].y,zfinal[i]);
	printf("\n");
*/
	
	/* Writing time elapsed */

	if(flag_save == 1)
	{	
	FILE *f6;
   	time_t t;
    	struct tm *tm;
    	 char date[30], *myDir;

    	t = time(NULL);
    	tm = localtime(&t);
    	strftime(date, sizeof(date)-1, "%d-%m-%Y_%H:%M:%S", tm);
    	static const char *directory = "ResultsKrigingTimeGPU-";

    	myDir = (char *)malloc(strlen(directory)+strlen(date)+1);
    	strcpy(myDir, directory);
    	strcat(myDir, date);
		
	f6 = fopen(myDir,"w");
	fprintf(f6,"Total time : %f s\n",milliseconds*0.001);

	fclose(f6);
	
	saveData(qp,zfinal,n);

	free(myDir);

	}

	// Free heap data
	hipsolverDnDestroy(start);
	hipblasDestroy(handlet);

	hipFree(kpd);
	hipFree(qpd);
	hipFree(zd);	
	hipFree(matricegpu);
	hipFree(zfinald);
	hipFree(Workspace);
	hipFree(devIpiv);
	hipFree(devInfo);
	
	free(kp);
	free(qp);
	free(Lwork);
	free(z);
	free(zfinal);	
	
	exit(0);	
}
